
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void add(int *a, int *b, int *c) {
  c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

#define N 512

int main(void) {
  int *a, *b, *c;
  int *d_a, *d_b, *d_c;
  int size = N * sizeof(int);

  hipMalloc((void **) &d_a, size);
  hipMalloc((void **) &d_b, size);
  hipMalloc((void **) &d_c, size);

  a = (int *)malloc(size);
  b = (int *)malloc(size);
  c = (int *)malloc(size);
  for (int i=0; i<N; i++) {
    a[i] = 2;
    b[i] = 7;
  }

  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  add<<<1,N>>>(d_a, d_b, d_c);

  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  for (int i=0; i<N; i++)
    printf("%d %d\n",i, c[i]);

  free(a); free(b); free(c);
  hipFree(d_a); hipFree(d_b); hipFree(d_c);

  return 0;
}