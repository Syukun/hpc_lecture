
#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdlib>
#include <cstdio>
#include <sys/time.h>

#define M 1024

double get_time() {
  struct timeval tv;
  gettimeofday(&tv, NULL);
  return double(tv.tv_sec)+double(tv.tv_usec)*1e-6;
}

__global__ void matmul(float *A, float *B, float *C, int N) {
  int i = blockIdx.y;
  int j = threadIdx.x + blockDim.x * blockIdx.x;
  float sum = 0.0f;
  __shared__ float s_A[M];
  for (int ks=0; ks<N; ks+=M) {
    __syncthreads();
    s_A[threadIdx.x] = A[N*i+ks+threadIdx.x];
    __syncthreads();
    for (int k=ks; k<ks+M; k++) {
      sum += s_A[k-ks] * B[N*k+j];
    }
  }
  C[N*i+j] = sum;
}

int main(int argc, char **argv) {
  int N = atoi(argv[1]);
  float * h_A = new float [N*N];
  float * h_B = new float [N*N];
  float * h_C = new float [N*N];
  float *d_A, *d_B, *d_C;
  int size = N * N * sizeof(float);
  hipMalloc((void **) &d_A, size);
  hipMalloc((void **) &d_B, size);
  hipMalloc((void **) &d_C, size);

  for (int i=0; i<N; i++) {
    for (int j=0; j<N; j++) {
      h_A[N*i+j] = drand48();
      h_B[N*i+j] = drand48();
      h_C[N*i+j] = 0;
    }
  }
  double tic = get_time();
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
  hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice);
  dim3 grid(N/M, N);
  matmul<<<grid,M>>>(d_A, d_B, d_C, N);
  hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
  hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);
  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

  double toc = get_time();
  printf("N=%d: %lf s (%lf GFlops)\n",N,toc-tic,2.*N*N*N/(toc-tic)/1e9);
  tic = get_time();
#pragma omp parallel for
  for (int i=0; i<N; i++) {
    for (int k=0; k<N; k++) {
      for (int j=0; j<N; j++) {
        h_C[N*i+j] -= h_A[N*i+k] * h_B[N*k+j];
      }
    }
  }
  toc = get_time();
  printf("N=%d: %lf s (%lf GFlops)\n",N,toc-tic,2.*N*N*N/(toc-tic)/1e9);
  float err = 0;
  for (int i=0; i<N; i++) {
    for (int j=0; j<N; j++) {
      err += fabs(h_C[N*i+j]);
    }
  }
  printf("error: %f\n",err/N/N);
  delete[] h_A;
  delete[] h_B;
  delete[] h_C;
}
